
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Function to get current time in seconds
double get_clock() {
    struct timeval tv;
    int ok;
    ok = gettimeofday(&tv, (void*)0);
    if (ok < 0) {
        printf("gettimeofday error");
    }
    return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

// GPU kernel for basic matrix multiplication
__global__ void gpuMatrixMultiply(double* A, double* B, double* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        double sum = 0.0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int sizes[] = {100, 200, 500, 1000, 2000};
    for (int idx = 0; idx < 5; idx++) {
        int N = sizes[idx];
        double *A, *B, *C, *d_A, *d_B, *d_C;

        // Allocate memory
        A = (double*)malloc(N * N * sizeof(double));
        B = (double*)malloc(N * N * sizeof(double));
        C = (double*)malloc(N * N * sizeof(double));
        hipMalloc(&d_A, N * N * sizeof(double));
        hipMalloc(&d_B, N * N * sizeof(double));
        hipMalloc(&d_C, N * N * sizeof(double));

        // Initialize matrices
        for (int i = 0; i < N * N; ++i) {
            A[i] = 1.0;
            B[i] = 1.0;
        }

        hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, N * N * sizeof(double), hipMemcpyHostToDevice);

        // Kernel configuration
        int threadsPerBlock = 16;
        dim3 threads(threadsPerBlock, threadsPerBlock);
        dim3 blocks((N + threadsPerBlock - 1) / threadsPerBlock, (N + threadsPerBlock - 1) / threadsPerBlock);

        // Perform matrix multiplication
        double t0 = get_clock();
        gpuMatrixMultiply<<<blocks, threads>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();
        double t1 = get_clock();

        printf("GPU Matrix Multiply - Size %d: %f s\n", N, t1 - t0);

        // Copy results back
        hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);

        // Free memory
        free(A);
        free(B);
        free(C);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }

    return 0;
}
